
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>
#include <stdio.h>
#define BLOCKSIZE 16
#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))
#define WARP_SIZE 32

using namespace nvcuda;

void init_matrices(half *a, half *b, float *c, int matsize) {
    for (int i = 0; i < matsize; ++i) {
        for (int j = 0; j < matsize; ++j) {
            a[i * matsize + j] = __float2half(1.0);
            b[i * matsize + j] = __float2half(1.0);
            c[i * matsize + j] = 1.0;
        }
    }
}

__global__ void mm_block_tc(int mat_size, half *A, half *B, float *C) {
    // Tile using a 2D grid
    int warpX = (blockIdx.x * blockDim.x + threadIdx.x) / WARP_SIZE;
    int warpY = (blockIdx.y * blockDim.y + threadIdx.y);
    
    // Declare the fragments
    wmma::fragment<wmma::matrix_a, BLOCKSIZE, BLOCKSIZE, BLOCKSIZE, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, BLOCKSIZE, BLOCKSIZE, BLOCKSIZE, half, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, BLOCKSIZE, BLOCKSIZE, BLOCKSIZE, float> acc_frag;
    wmma::fragment<wmma::accumulator, BLOCKSIZE, BLOCKSIZE, BLOCKSIZE, float> c_frag;

    // Initialize the output to zero
    wmma::fill_fragment(acc_frag, 0.0f);

    // Loop over the K dimension
    for (int i = 0; i < mat_size; i += BLOCKSIZE) {
        // Bounds checking
        int aRow = warpX * BLOCKSIZE;
        int aCol = i;
        int bRow = i;
        int bCol = warpY * BLOCKSIZE;

        // Load the inputs
        wmma::load_matrix_sync(a_frag, A + aRow * mat_size + aCol, mat_size);
        wmma::load_matrix_sync(b_frag, B + bRow * mat_size + bCol, mat_size);

        // Perform the matrix multiplication
        wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
    }

    // Load in the current value of c, scale it by beta, and add this our result scaled by alpha
    int cRow = warpX * BLOCKSIZE;
    int cCol = warpY * BLOCKSIZE;

    wmma::load_matrix_sync(c_frag, C + cRow * mat_size + cCol, mat_size, wmma::mem_row_major);

    for(int i=0; i < c_frag.num_elements; i++) {
        c_frag.x[i] = acc_frag.x[i];
    }

    // Store the output
    wmma::store_matrix_sync(C + cRow * mat_size + cCol, c_frag, mat_size, wmma::mem_row_major);
}


__global__ void mm_naive(int mat_size,  half *A, half *B, float *C) {
  // compute position in C that this thread is responsible for
  const uint x = blockIdx.x * blockDim.x + threadIdx.x;
  const uint y = blockIdx.y * blockDim.y + threadIdx.y;

  // `if` condition is necessary for when M or N aren't multiples of 32.
  if (x < mat_size && y < mat_size) {
    float tmp = 0.0;
    for (int i = 0; i < mat_size; ++i) {
      tmp += (float)A[x * mat_size + i] * (float)B[i * mat_size + y];
    }

    C[x * mat_size + y] = tmp ;
  }
}

void run_matrix_multiplication(int N) {
    printf("\nRunning matrix multiplication for size %d x %d\n", N, N);
    
    half *mat_a, *mat_b;
    float *mat_c;

    mat_a = (half*)malloc(N * N * sizeof(half));
    mat_b = (half*)malloc(N * N * sizeof(half));
    mat_c = (float*)malloc(N * N * sizeof(float));

    init_matrices(mat_a, mat_b, mat_c, N);

    half *d_mat_a, *d_mat_b;
    float *d_mat_c;
    hipMalloc(&d_mat_a, N * N * sizeof(half));
    hipMalloc(&d_mat_b, N * N * sizeof(half));
    hipMalloc(&d_mat_c, N * N * sizeof(float));

    hipMemcpy(d_mat_a, mat_a, N * N * sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(d_mat_b, mat_b, N * N * sizeof(half), hipMemcpyHostToDevice);
   
    dim3 gridDim;
    dim3 blockDim; 
    blockDim.x = 32;
    blockDim.y = 1;
    gridDim.x = CEIL_DIV(N, BLOCKSIZE * blockDim.x / 32); 
    gridDim.y = CEIL_DIV(N, BLOCKSIZE * blockDim.y);

    hipEvent_t start1, stop1, start2, stop2;
    float milliseconds = 0;

    // Warm up
    mm_naive<<<gridDim, blockDim>>>(N, d_mat_a, d_mat_b, d_mat_c);
    hipDeviceSynchronize();

    // TensorCore version
    hipEventCreate(&start2);
    hipEventCreate(&stop2);
    hipEventRecord(start2);
    mm_block_tc<<<gridDim, blockDim>>>(N, d_mat_a, d_mat_b, d_mat_c);
    hipEventRecord(stop2);
    hipEventSynchronize(stop2);
    hipEventElapsedTime(&milliseconds, start2, stop2);
    printf("Time taken with TensorCore: %f ms\n", milliseconds);

    // Naive version
    blockDim.x = 32;
    blockDim.y = 1;
    gridDim.x = CEIL_DIV(N, blockDim.x / 32); 
    gridDim.y = CEIL_DIV(N, blockDim.y);

    hipEventCreate(&start1);
    hipEventCreate(&stop1);
    hipEventRecord(start1);
    mm_naive<<<gridDim, blockDim>>>(N, d_mat_a, d_mat_b, d_mat_c);
    hipEventRecord(stop1);
    hipEventSynchronize(stop1);
    hipEventElapsedTime(&milliseconds, start1, stop1);
    printf("Time taken without TensorCore: %f ms\n", milliseconds);
    
    hipMemcpy(mat_c, d_mat_c, N * N * sizeof(float), hipMemcpyDeviceToHost);  

    free(mat_a);
    free(mat_b);
    free(mat_c);
    hipFree(d_mat_a);
    hipFree(d_mat_b);
    hipFree(d_mat_c);
}

int main() {
    int sizes[] = {256, 512, 1024, 2048, 4096};
    int num_sizes = sizeof(sizes) / sizeof(sizes[0]);

    for (int i = 0; i < num_sizes; i++) {
        run_matrix_multiplication(sizes[i]);
        run_matrix_multiplication(sizes[i]);
        run_matrix_multiplication(sizes[i]);
        run_matrix_multiplication(sizes[i]);
        run_matrix_multiplication(sizes[i]);
    }
    
    return 0;
}